#include "hip/hip_runtime.h"
/*!
 * \file cuda_multicorrelator.cc
 * \brief High optimized CUDA vector multiTAP correlator class
 * \authors <ul>
 *          <li> Javier Arribas, 2015. jarribas(at)cttc.es
 *          </ul>
 *
 * Class that implements a high optimized vector multiTAP correlator class for CUDAs
 *
 * -------------------------------------------------------------------------
 *
 * Copyright (C) 2010-2015  (see AUTHORS file for a list of contributors)
 *
 * GNSS-SDR is a software defined Global Navigation
 *          Satellite Systems receiver
 *
 * This file is part of GNSS-SDR.
 *
 * GNSS-SDR is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * GNSS-SDR is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with GNSS-SDR. If not, see <http://www.gnu.org/licenses/>.
 *
 * -------------------------------------------------------------------------
 */

#include "cuda_multicorrelator_real_codes.h"
#include "cuda/cuda_kernels.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <gnuradio/gr_complex.h>


cuda_multicorrelator_real_codes::cuda_multicorrelator_real_codes()
{
    cu_sig_in = nullptr;
    cu_local_code_in = nullptr;
    cu_shifts_chips = nullptr;
    cu_corr_out = nullptr;
    cu_local_codes_resampled = nullptr;
    d_code_length_chips = 0;
    d_n_correlators = 0;
}


cuda_multicorrelator_real_codes::~cuda_multicorrelator_real_codes()
{
    if (cu_local_codes_resampled != nullptr)
    {
        this->free();
    }
}


bool cuda_multicorrelator_real_codes::init(int device, int max_signal_length_samples, int code_length_chips, int n_correlators)
{
    d_n_correlators = n_correlators;
    cu_selected_device = device;
    gpuErrchk(hipSetDevice(cu_selected_device));
    cu_num_threads = 128;
    cu_num_blocks = (int)pow(2, ceil(log2((double)max_signal_length_samples))) / cu_num_threads;
    // ALLOCATE MEMORY FOR INTERNAL vectors
    size_t size = max_signal_length_samples * sizeof(hipComplex);
    gpuErrchk(hipMalloc((void**)&cu_sig_in, size));
    gpuErrchk(hipMalloc((void**)&cu_corr_out, n_correlators * sizeof(hipComplex)));
    gpuErrchk(hipMalloc((void**)&cu_local_code_in, code_length_chips * sizeof(float)));
    gpuErrchk(hipMalloc((void**)&cu_shifts_chips, n_correlators * sizeof(float)));
    gpuErrchk(hipMalloc((void**)&cu_red_tmp, n_correlators * cu_num_blocks * sizeof(hipComplex)));
//    d_local_codes_resampled = static_cast<std::complex<float>**>(volk_gnsssdr_malloc(n_correlators * sizeof(std::complex<float>*), volk_gnsssdr_get_alignment()));

    gpuErrchk(hipMalloc((void**)&cu_local_codes_resampled, size * n_correlators));
    gpuErrchk(hipMalloc((void**)&cu_phase, size));
    return true;
}


bool cuda_multicorrelator_real_codes::set_local_code_and_taps(int code_length_chips, const float *local_code_in,
                                                              float *shifts_chips, int n_correlations)
{
//    cu_local_code_in = local_code_in;
    gpuErrchk(hipMemcpy(cu_local_code_in, local_code_in, sizeof(float) * code_length_chips, hipMemcpyHostToDevice));
//    d_shifts_chips = shifts_chips;
    gpuErrchk(hipMemcpy(cu_shifts_chips, shifts_chips, sizeof(float) * n_correlations, hipMemcpyHostToDevice));
    d_code_length_chips = code_length_chips;
    return true;
}


bool cuda_multicorrelator_real_codes::set_input_output_vectors(gr_complex* corr_out, hipComplex* sig_in)
{
    // Save CUDA pointers
    cu_sig_in = sig_in;
    d_corr_out = corr_out;
    return true;
}


void cuda_multicorrelator_real_codes::update_local_code(int correlator_length_samples, float rem_code_phase_chips, float code_phase_step_chips)
{
    gpuErrchk(hipSetDevice(cu_selected_device));
    cuda_xn_resampler_xn<<<cu_num_blocks, cu_num_threads>>>(cu_local_codes_resampled,
                                                            cu_local_code_in,
                                                            rem_code_phase_chips,
                                                            code_phase_step_chips,
                                                            cu_shifts_chips,
                                                            d_code_length_chips,
                                                            d_n_correlators,
                                                            correlator_length_samples);
    gpuErrchk(hipGetLastError());
}


bool cuda_multicorrelator_real_codes::Carrier_wipeoff_multicorrelator_resampler(
        float rem_carrier_phase_in_rad,
        float phase_step_rad,
        float rem_code_phase_chips,
        float code_phase_step_chips,
        int signal_length_samples)
{
    gpuErrchk(hipSetDevice(cu_selected_device));
//    gpuErrchk(hipMemcpy(cu_sig_in, d_sig_in, sizeof(hipComplex) * signal_length_samples, hipMemcpyHostToDevice));
    update_local_code(signal_length_samples, rem_code_phase_chips, code_phase_step_chips);
    // Regenerate phase at each call in order to avoid numerical issues
    cuda_sincos<<<cu_num_blocks, cu_num_threads>>>(cu_phase, -phase_step_rad, -rem_carrier_phase_in_rad, signal_length_samples);
    gpuErrchk(hipGetLastError());

    // call CUDA kernel
    cuda_x2_dot_prod_xn_stage1<<<cu_num_blocks, cu_num_threads, cu_num_threads * sizeof(hipComplex) * d_n_correlators>>>(
            cu_red_tmp, cu_sig_in, cu_phase, cu_local_codes_resampled, d_n_correlators, signal_length_samples);
    gpuErrchk(hipGetLastError());

    cuda_x2_dot_prod_xn_stage2<<<1, cu_num_blocks>>>(cu_corr_out, cu_red_tmp, d_n_correlators, signal_length_samples);
    gpuErrchk(hipGetLastError());
//    gpuErrchk(hipMemcpy(d_corr_out, cu_corr_out, sizeof(hipComplex) * d_n_correlators, hipMemcpyDeviceToHost));
    return true;
}


bool cuda_multicorrelator_real_codes::free()
{
    gpuErrchk(hipSetDevice(cu_selected_device));
    gpuErrchk(hipFree(cu_red_tmp));
    // Free memory
    if (cu_local_codes_resampled != nullptr)
    {
        gpuErrchk(hipFree(cu_local_codes_resampled));
        cu_local_codes_resampled = nullptr;
    }
    return true;
}
